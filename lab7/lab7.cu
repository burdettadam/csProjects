
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <math.h>
#define MAXIT 360
#define N 1024
#define M 1024
int *lkeepgoing;
float *iplate;
float *oplate;
float *tmp;

/* Return the current time in seconds, using a double precision number.       */
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}

__global__ void kernel(float * d_matrix, size_t pitch) {
    int colsPerThread = 1;//32 threads per block ,256 cells in block-> 256/32
    int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
    for (int j = rowstart; j < rowstart+colsPerThread; j ++) {
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
        int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
        for (int i = colstart; i < colstart + colsPerThread; i ++) {
            row_d_matrix[i] = j * M + i;
        }
    }
}

__global__ void fill(float * iplate, size_t ipitch,float * oplate, size_t opitch) {
    int index;
    int colsPerThread = 1;//32 threads per block ,256 cells in block-> 256/32
    int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
    for (int j = rowstart ; j < rowstart+colsPerThread; j ++) {

        float* row_iplate = (float*)((char*)iplate + j*ipitch);
        float* row_oplate = (float*)((char*)oplate + j*opitch);

        int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
        for (int i = colstart; i < colstart + colsPerThread; i ++) {
           // row_iplate[i] = (j * M + i) + (j * M + i);
            index = j * M + i;
            if (index <= M || (index % (M-1)) == 0){
                row_iplate[i] = 0.0;
                row_oplate[i] = 0.0;
            }
            else if (index >= 67100672 || index == (200 * M + 500) || (index < (400*M+331) && index > (400 * M +0))){// might be one off
                row_iplate[i] = 100.0;
                row_oplate[i] = 100.0;
            }
            else{
                row_iplate[i] = 50.0;
                row_oplate[i] = 50.0;
            }
        }
    }
}
__global__ void doCalc(float * iplate, size_t ipitch,float * oplate, size_t opitch) {
    int index;
    int colsPerThread = 1;//32 threads per block ,256 cells in block-> 256/32
    float* n_row_oplate;
    float* s_row_oplate;
    int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
    for (int j = rowstart ; j < rowstart+colsPerThread; j ++) {

        float* row_iplate = (float*)((char*)iplate + j*ipitch);
        float* row_oplate = (float*)((char*)oplate + j*opitch);
        if (j == 0 || j == N){
            continue;
        }
        else {
            n_row_oplate = (float*)((char*)oplate + (j-1)*opitch);
            s_row_oplate = (float*)((char*)oplate + (j+1)*opitch);
        }

        int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
        for (int i = colstart; i < colstart + colsPerThread; i ++) {
           // row_iplate[i] = (j * M + i) + (j * M + i);
            index = j * M + i;
            if (i == 0 || i == M){
                continue;
            }
            if (index >= 67100672 || index == (200 * M + 500) || (index < (400*M+331) && index > (400 * M +0))){// might be one off
                row_iplate[i] = 100.0;
            }
            else{
                row_iplate[i] = (( s_row_oplate[i]//bottom
                             + n_row_oplate[i]//top
                             + row_oplate[i+1]//right
                             + row_oplate[i-1])//left
                            + (4.0 * row_oplate[i])) / 8.0;
            }
        }
    }
}
__global__ void doCheck(float * iplate, size_t ipitch,float * oplate, size_t opitch,int *lkeepgoing,size_t lpitch) {
    float delta = 0.0;
    //int index;
    int colsPerThread = 1;//32 threads per block ,256 cells in block-> 256/32
    int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
    for (int j = rowstart ; j < rowstart+colsPerThread; j ++) {

        float* row_iplate = (float*)((char*)iplate + j*ipitch);
        float* row_oplate = (float*)((char*)oplate + j*opitch);
        float* row_lplate = (float*)((char*)lkeepgoing + j*lpitch);
        if (j == 0 || j == N){
            continue;
        }
        int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
        for (int i = colstart; i < colstart + colsPerThread; i ++) {
           // index = j * M + i;
            if (i == 0 || i == M){
                continue;
            }
            delta = fabs((row_iplate[i] - row_oplate[i]));
            if (delta > 0.0500) {
                row_lplate[i] = 1; // keep going
            }else{
                row_lplate[i] = 0; // steady state
            }
        }
    }
}

__global__ void reduceSum(int *lkeepgoing, size_t lpitch, int *odata) {
    int colsPerThread = 1;//32 threads per block ,256 cells in block-> 256/32
    if(threadIdx.x == 0) {
        odata[blockIdx.x] = 0;
        int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
        for (int j = rowstart ; j < rowstart+blockDim.y; j ++) {
            
            float* row_lplate = (float*)((char*)lkeepgoing + j*lpitch);
            if (j == 0 || j == N){
                continue;
            }
            int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
            for (int i = colstart; i < colstart + blockDim.x; i ++) {
                odata[blockIdx.x] +=  row_lplate[i];
            }
        }
    }
   
}
__global__ void reduce1(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
        sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void reduce2(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void reduce3(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s = (blockDim.x/2) ; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {


    //h_matrix = (float *) malloc(M * N * sizeof(float));
    //dc_matrix = (float *) malloc(M * N * sizeof(float));
    double t0, tottime, start = When();
    size_t ipitch;
    size_t opitch;
    size_t lpitch;
    hipMallocPitch(&iplate, &ipitch, M * sizeof(float), N);
    hipMallocPitch(&oplate, &opitch, M * sizeof(float), N);
    hipMallocPitch(&lkeepgoing, &lpitch, M * sizeof(float), N);
   // cudaMallocPitch(&d_matrix, &pitch, M * sizeof(float), N);

    dim3 threadsPerBlock(32, 32, 1); // number of threads per block 
    dim3 numBlocks(N/threadsPerBlock.x,M/threadsPerBlock.y, 1); // number of blocks in grid 16x16
    fill<<<numBlocks, threadsPerBlock>>>(iplate, ipitch, oplate, opitch);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
        return 0;
    }
    int *keepgoing_single;
    int *keepgoing_sums;
    int keepgoing;
    int iteration;
    hipMalloc((void **)&keepgoing_single, 1 * sizeof(int));
    keepgoing = 1;
    hipMalloc((void **)&keepgoing_sums, M * sizeof(int));
    float totaltime = 0.0;
    for (iteration = 0; (iteration < MAXIT) && keepgoing; iteration++)
    {
        doCalc<<< numBlocks, threadsPerBlock >>>(iplate, ipitch, oplate, opitch);
        hipDeviceSynchronize();
        error = hipGetLastError();
        if(error != hipSuccess) {
            printf("%s\n",hipGetErrorString(error));
            return 0;
        }
        doCheck<<< numBlocks, threadsPerBlock >>>(iplate, ipitch, oplate, opitch, lkeepgoing, lpitch);
        hipDeviceSynchronize();
        error = hipGetLastError();
        if(error != hipSuccess) {
            printf("%s\n",hipGetErrorString(error));
            return 0;
        }
        reduceSum<<< numBlocks, threadsPerBlock>>>(lkeepgoing,lpitch, keepgoing_sums );
        hipDeviceSynchronize();
        error = hipGetLastError();
        if(error != hipSuccess) {
            printf("%s\n",hipGetErrorString(error));
            return 0;
        }

        // Now we have the sum for each row in the first column, 
        //  reduce to one value
        t0 = When();
        int timeit;
        for(timeit = 0; timeit < 10000; timeit++){
            //reduce1<<<1, 1024, 1024*sizeof(int)>>>(keepgoing_sums, keepgoing_single);
            //reduce2<<<1, 1024, 1024*sizeof(int)>>>(keepgoing_sums, keepgoing_single);
            reduce3<<<1, 1024, 1024*sizeof(int)>>>(keepgoing_sums, keepgoing_single);
            hipDeviceSynchronize();
            error = hipGetLastError();
            if(error != hipSuccess) {
                printf("%s\n",hipGetErrorString(error));
                return 0;
            }
        }
        tottime = When()-t0;
        
        keepgoing = 0;
        hipMemcpy(&keepgoing, keepgoing_single, 1 * sizeof(int), hipMemcpyDeviceToHost);
        totaltime += (tottime/10000);
        /* swap the new value pointer with the old value pointer */
        tmp = oplate;
        oplate = iplate;
        iplate = tmp;
    }
    totaltime = totaltime/ iteration;
    hipFree(iplate); 
    hipFree(oplate); 
    hipFree(keepgoing_single); 
    hipFree(keepgoing_sums);
    printf("Finished in %d iterations at %f, with reduce average time in %f.\n", iteration,When()-start,totaltime);
}
