
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>

#define BLOCKSIZE 1024
#define MAXIT 1
#define TOTROWS		(BLOCKSIZE*8)
#define TOTCOLS		(BLOCKSIZE*8)
#define NOTSETLOC       -1 // for cells that are not fixed

#define QMAX(x,y) (((x) > (y))? (x): (y))


float *iplate;
float *host_matrix;
int ncols, nrows;

void Compute();


int main(int argc, char *argv[])
{
	ncols = TOTCOLS;
	nrows = TOTROWS;
	hipMalloc((void **) &iplate, nrows * ncols * sizeof(float));
	Compute();

	return 0;
}

__global__ void InitArrays(float *ip, int ncols)
{
	int i;
	// ncols is the same as number of threads
        // Each block gets a row, each thread will fill part of a row
	// Calculate the offset of the row
    int blockOffset = blockIdx.x * ncols; // first possion in array of the block3
        // The number of cols per thread
    int colsPerThread = ncols/blockDim.x;
        // Calculate our offset into the row for the thread
	int colStartPos = threadIdx.x * (colsPerThread); // col index 
	// position = arrayaddress + position of block + position of thread
	//int col = threadIdx.x + blockIdx.x * colsPerThread ;//* blockDim.x // I thick this is correct..
	int col = colStartPos;
	int row = blockIdx.x ;
	float *ippos = ip + blockOffset+ colStartPos;

	for (i = 0; i < colsPerThread; i++) {
		ippos[i] = i;
	}
}
void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        printf("%2f ",d[i]);
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
void Compute()
{
	int blocksize = BLOCKSIZE;
	ncols = TOTCOLS;
	nrows = TOTROWS;
	host_matrix = (float *) malloc(ncols * nrows * sizeof(float));
    for (int j = 0; j < nrows; j++) {
        for (int i = 0; i < ncols; i++) {
            host_matrix[j * ncols + i] = (j * ncols + i);
        }
    }
    //cudaMalloc((void**)&d_A, (N*N)*sizeof(float));
    //cudaMemcpy(d_A, A, (N*N)*sizeof(float), cudaMemcpyHostToDevice);
	// One block per row
	InitArrays<<< nrows, blocksize >>>(iplate, ncols);
	  // print out the result element by element
	  for(int row = 0; row < TOTROWS; ++row){
	    for(int col = 0; col < TOTCOLS; ++col){
	      printf("%2f ", iplate[row * TOTROWS + col]);
	    }
	    printf("\n");
	  }
	  printf("\n");
	 verify(host_matrix, iplate, TOTCOLS * TOTROWS);
}

