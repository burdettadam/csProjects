
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>

#define BLOCKSIZE 1024
#define TOTROWS		(BLOCKSIZE*8)
#define TOTCOLS		(BLOCKSIZE*8)

float *iplate;
float *host_matrix;
int ncols, nrows;

__global__ void InitArrays(float *ip){
	int i;
	// ncols is the same as number of threads
        // Each block gets a row, each thread will fill part of a row
	// Calculate the offset of the row
    int blockOffset = blockIdx.x * TOTCOLS; // first possion in array of the block3
        // The number of cols per thread
    int colsPerThread = TOTCOLS/blockDim.x;
        // Calculate our offset into the row for the thread
	int colStartPos = threadIdx.x * (colsPerThread); // col index 
	// position = arrayaddress + position of block + position of thread
	//int col = threadIdx.x + blockIdx.x * colsPerThread ;//* blockDim.x // I thick this is correct..
	//int col = colStartPos;
	//int row = blockIdx.x ;

	float *ippos = ip + blockOffset+ colStartPos;

	for (i = 0; i < colsPerThread; i++) {
		ippos[i] = i;
	}
}
void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        printf("%2f ",d[i]);
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
void Compute()
{
	int blocksize = BLOCKSIZE;
	ncols = TOTCOLS;
	nrows = TOTROWS;
	host_matrix = (float *) malloc(ncols * nrows * sizeof(float));
    for (int j = 0; j < nrows; j++) {
        for (int i = 0; i < ncols; i++) {
            host_matrix[j * ncols + i] = (j * ncols + i);
        }
    }
    //cudaMemcpy(d_A, A, (N*N)*sizeof(float), cudaMemcpyHostToDevice);
	// One block per row
	InitArrays<<< nrows, blocksize >>>(iplate);
	  // print out the result element by element
	  for(int row = 0; row < TOTROWS; ++row){
	    for(int col = 0; col < TOTCOLS; ++col){
	      printf("%2f ", iplate[row * TOTROWS + col]);
	    }
	    printf("\n");
	  }
	  printf("\n");
	 verify(host_matrix, iplate, TOTCOLS * TOTROWS);
}

int main(int argc, char *argv[])
{
	ncols = TOTCOLS;
	nrows = TOTROWS;
	hipMalloc((void **) &iplate, nrows * ncols * sizeof(float));
	Compute();

	return 0;
}

