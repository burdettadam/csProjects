#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCKSIZE 1024
#define MAXIT 1
#define TOTROWS		(BLOCKSIZE*8)
#define TOTCOLS		(BLOCKSIZE*8)
#define NOTSETLOC       -1 // for cells that are not fixed

#define QMAX(x,y) (((x) > (y))? (x): (y))


int *lkeepgoing;
float *iplate;
float *oplate;
float *fixed;
float *tmp;
int ncols, nrows;

double When();
void Compute();


int main(int argc, char *argv[])
{
	double start;
	ncols = TOTCOLS;
	nrows = TOTROWS;

	hipMalloc((void **) &lkeepgoing, nrows * ncols * sizeof(int));
	hipMalloc((void **) &iplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &oplate, nrows * ncols * sizeof(float));
	hipMalloc((void **) &fixed,  nrows * ncols * sizeof(float));
	fprintf(stderr,"Memory allocated\n");

	start = When();
	/* Now proceed with the Jacobi algorithm */
	Compute();

	printf("Total Time is: %lf sec.\n", (When() - start));

	return 0;
}

__global__ void InitArrays(float *ip, float *op, float *fp, int *kp, int ncols)
{
	int i;
	// ncols is the same as number of threads
        // Each block gets a row, each thread will fill part of a row
	// Calculate the offset of the row
    int blockOffset = blockIdx.x * ncols; // first possion in array of the block3
        // The number of cols per thread
    int colsPerThread = ncols/blockDim.x;
        // Calculate our offset into the row for the thread
	int colStartPos = threadIdx.x * (colsPerThread); // col index 
	// position = arrayaddress + position of block + position of thread
	//int col = threadIdx.x + blockIdx.x * colsPerThread ;//* blockDim.x // I thick this is correct..
	int col = colStartPos;
	int row = blockIdx.x ;
	float *ippos = ip + blockOffset+ colStartPos;
	float *fppos = fp + blockOffset+ colStartPos;
	float *oppos = op + blockOffset+ colStartPos;
	int *kppos = kp + blockOffset+ colStartPos;

	for (i = 0; i < colsPerThread; i++) {
		fppos[i] = NOTSETLOC; // Not Fixed
		ippos[i] = 50;
		oppos[i] = 50;
	    kppos[i] = 1; // Keep Going
	}
        // set the rest of the boundary and fixed positions
	for (i = 0; i < colsPerThread; i++) {
		col += i;
        if (row == 0 || col == 0 || col == ncols-1 ){
            ippos[i] = 0.0;
            oppos[i] = 0.0;
        }
        else if (row == (TOTCOLS - 1) ){
            ippos[i] = 100.0;
            oppos[i] = 100.0;
        }
	}
	if (row == 400 ){
		col = colStartPos;
	    for ( i = 0; i < colsPerThread; i++) {
			col += i;
	    	if (col < 331){
		        ippos[i]=100.0;
		        oppos[i]=100.0;
		    }else{
		    	break;
		    }
	    }

	}
	else if (row == 200){
		col = colStartPos;
		if ( col <= 500 && (col + ncols) >= 500 ){
			ippos[500]=100.0;
	    	oppos[500]=100.0;	
		}
		
	}
}
__global__ void doCalc(float *iplate, float *oplate, int ncols)
{//This is called non-uniform indexing
	/* Compute the 5 point stencil for my region *///??????????????
	int i;
	// ncols is the same as number of threads
        // Each block gets a row, each thread will fill part of a row
	// Calculate the offset of the row
    int blockOffset = blockIdx.x * ncols; // first possion in array of the block3
        // The number of cols per thread
    int colsPerThread = ncols/blockDim.x;
        // Calculate our offset into the row for the thread
	int colStartPos = threadIdx.x * (colsPerThread); // col index 
	// position = arrayaddress + position of block + position of thread
	//int col = threadIdx.x + blockIdx.x * colsPerThread ;//* blockDim.x // I thick this is correct..
	int col = colStartPos;
	int row = blockIdx.x ;
	int rowup,rowdown;
	if(blockIdx.x == 0){
		rowup = 0;
	}else{
		rowup = blockIdx.x - 1 ;
	}
	if(blockIdx.x == ncols-1){
		return;
	}
	rowdown = blockIdx.x + 1;

	float *ippos = iplate + blockOffset+ colStartPos;
	float *oppos = oplate + blockOffset+ colStartPos;
	for (i = 0; i < colsPerThread; i++) {
		col += i;
		if (col == 0 || col == ncols){
			continue;
		}else{
 			ippos[i] =(( oppos[rowdown][col]//bottom
                             + oppos[rowup[col]//top
                             + oppos[row][col+1]//right
                             + oppos[row][col-1])//left
                            + (4.0 * oppos[row][col])) / 8.0;
		}
  	}
    if (row == 400 ){
		col = colStartPos;
	    for ( i = 0; i < colsPerThread; i++) {
			col += i;
	    	if (col < 331){
		        ippos[i]=100.0;
		        oppos[i]=100.0;
		    }else{
		    	break;
		    }
	    }

	}
	else if (row == 200){
		col = colStartPos;
		if ( col <= 500 && (col + ncols) >= 500 ){
			ippos[500]=100.0;
    		oppos[500]=100.0;
		}

	}
}

__global__ void doCheck(float *iplate, float *oplate, float *fixed, int *lkeepgoing, int ncols)
{
	// Calculate keepgoing array
		int i,col;
	// ncols is the same as number of threads
        // Each block gets a row, each thread will fill part of a row
	// Calculate the offset of the row
    int blockOffset = blockIdx.x * ncols; // first possion in array of the block3
        // The number of cols per thread
    int colsPerThread = ncols/blockDim.x;
        // Calculate our offset into the row for the thread
	int colStartPos = threadIdx.x * (colsPerThread); // col index 
	// position = arrayaddress + position of block + position of thread
	//int col = threadIdx.x + blockIdx.x * colsPerThread ;//* blockDim.x // I thick this is correct..
	int col = colStartPos;
	int row = blockIdx.x ;
	float *ippos = iplate + blockOffset+ colStartPos;
	float *fppos = fixed + blockOffset+ colStartPos;
	float *oppos = oplate + blockOffset+ colStartPos;
	int kppos = lkeepgoing + blockOffset+ colStartPos;
	float delta = 0.0;
    float averageNabor =0.0;
    for (i = 0; i < colsPerThread; i++) {
		col += i;
		if (col == 0 || col == ncols){
			continue;
		}else{
		    delta = fabs((ippos[col] - oppos[col]));
            if (delta > 0.0500) {
                kppos[col] = 1; // keep going
            }else{
            	kppos[col] = 0; // steady state
            }
        }
	}
}

__global__ void reduceSingle(int *idata, int *single, int nrows)
{
	// Reduce rows to the first element in each row
	int i;
        int rowStartPos;
        int colsPerThread;
	extern __shared__ int parts[];
	
        // Each block gets a row, each thread will reduce part of a row

        // Calculate our offset into the row
	rowStartPos = threadIdx.x * (nrows/blockDim.x);
        // The number of cols per thread
        colsPerThread = nrows/blockDim.x;

	// Sum my part of one dimensional array and put it shared memory
	parts[threadIdx.x] = 0;
	for (i = threadIdx.x; i < nrows; i+=blockDim.x) {
		parts[threadIdx.x] += idata[i];
	}
	int tid = threadIdx.x;
        if (tid < 512) { parts[tid] += parts[tid + 512];}  
        __syncthreads();
        if (tid < 256) { parts[tid] += parts[tid + 256];}
        __syncthreads();
        if (tid < 128) { parts[tid] += parts[tid + 128];}
        __syncthreads();
        if (tid < 64) { parts[tid] += parts[tid + 64];}
        __syncthreads();
        if (tid < 32) { parts[tid] += parts[tid + 32];}
        __syncthreads();
	if(threadIdx.x == 0) {
		*single = 0;
		for(i = 0; i < 32; i++) {
			*single += parts[i];
		}
	}
}
__global__ void reduceSum(int *idata, int *odata, unsigned int ncols)
{
	// Reduce rows to the first element in each row
	int i;
        int blockOffset;
        int rowStartPos;
        int colsPerThread;
        int *mypart;
	
        // Each block gets a row, each thread will reduce part of a row

	// Calculate the offset of the row
        blockOffset = blockIdx.x * ncols;
        // Calculate our offset into the row
	rowStartPos = threadIdx.x * (ncols/blockDim.x);
        // The number of cols per thread
        colsPerThread = ncols/blockDim.x;

	mypart = idata + blockOffset + rowStartPos;

	// Sum all of the elements in my thread block and put them 
        // into the first column spot
	for (i = 1; i < colsPerThread; i++) {
		mypart[0] += mypart[i];
	}
	__syncthreads(); // Wait for everyone to complete
        // Now reduce all of the threads in my block into the first spot for my row
	if(threadIdx.x == 0) {
		odata[blockIdx.x] = 0;
		for(i = 0; i < blockDim.x; i++) {
			odata[blockIdx.x] += mypart[i*colsPerThread];
		}
	}
	// We cant synchronize between blocks, so we will have to start another kernel
}
	
void Compute()
{
	int *keepgoing_single;
	int *keepgoing_sums;
	int keepgoing;
	int blocksize = BLOCKSIZE;
	int iteration;
	double t0, tottime;

	ncols = TOTCOLS;
	nrows = TOTROWS;

	// One block per row
	InitArrays<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols);
	hipMalloc((void **)&keepgoing_single, 1 * sizeof(int));
	keepgoing = 1;
	hipMalloc((void **)&keepgoing_sums, nrows * sizeof(int));
 	int *peek = (int *)malloc(nrows*sizeof(int));

	for (iteration = 0; (iteration < MAXIT) && keepgoing; iteration++)
	{
		doCalc<<< nrows, blocksize >>>(iplate, oplate, ncols);
		doCheck<<< nrows, blocksize >>>(iplate, oplate, fixed, lkeepgoing, ncols);
		reduceSum<<< nrows, blocksize>>>(lkeepgoing, keepgoing_sums, ncols);
		//reduce1<<<nrows, blocksize>>>((lkeepgoing, keepgoing_sums, ncols));
		//reduce2<<<nrows, blocksize>>>((lkeepgoing, keepgoing_sums, ncols));
		//reduce3<<<nrows, blocksize>>>((lkeepgoing, keepgoing_sums, ncols));
		hipMemcpy(peek, keepgoing_sums, nrows*sizeof(int), hipMemcpyDeviceToHost);
//		fprintf(stderr, "after hipMemcpy \n");
//		int i;
 //		for(i = 0; i < nrows; i++) {
//			fprintf(stderr, "%d, ",peek[i]);
//		}
		// Now we have the sum for each row in the first column, 
		//  reduce to one value
		t0 = When();
 		int timeit;
		for(timeit = 0; timeit < 10000; timeit++){
			reduceSingle<<<1, blocksize, blocksize*sizeof(int)>>>(keepgoing_sums, keepgoing_single, nrows);
		}
		tottime = When()-t0;
		
		keepgoing = 0;
		hipMemcpy(&keepgoing, keepgoing_single, 1 * sizeof(int), hipMemcpyDeviceToHost);
		fprintf(stderr, "keepgoing = %d time %f\n", keepgoing, tottime);

		/* swap the new value pointer with the old value pointer */
		tmp = oplate;
		oplate = iplate;
		iplate = tmp;
	}
	free(peek);
	hipFree(keepgoing_single);
	hipFree(keepgoing_sums);
	fprintf(stderr,"Finished in %d iterations\n", iteration);
}
__global__ void reduce1(int *g_idata, int *g_odata,int nrows) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	int rowStartPos = threadIdx.x * (nrows/blockDim.x);
    int colsPerThread = nrows/blockDim.x;
	unsigned int i = blockIdx.x*colsPerThread + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=1; s < colsPerThread; s *= 2) {
		if (tid % (2*s) == 0) {
		sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void reduce2(int *g_idata, int *g_odata,int nrows) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	int rowStartPos = threadIdx.x * (nrows/blockDim.x);
    int colsPerThread = nrows/blockDim.x;
	unsigned int i = blockIdx.x*colsPerThread + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=1; s < colsPerThread; s *= 2) {
		int index = 2 * s * tid;
		if (index < colsPerThread) {
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void reduce3(int *g_idata, int *g_odata,int nrows) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	int rowStartPos = threadIdx.x * (nrows/blockDim.x);
    int colsPerThread = nrows/blockDim.x;
	unsigned int i = blockIdx.x*colsPerThread + threadIdx.x;
	sdata[tid] = g_idata[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=colsPerThread/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
/* Return the current time in seconds, using a double precision number.       */
double When()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}
