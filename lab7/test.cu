/*#include <stdio.h>
#include <assert.h>
#define row 22
#define col 22
__global__ void kernel(float * device_matrix, size_t pitch) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < row; j += blockDim.y * gridDim.y) {
        float* row_device_matrix = (float*)((char*)device_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < col; i += blockDim.x * gridDim.x) {
            row_device_matrix[i] = (j * col + i) + (j * col + i);
        }
    }
}
void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        printf("%2f ",d[i]);
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
int main() {
    float *host_matrix;
    float *device_matrix;
    float *deviceCopy_matrix;
    host_matrix = (float *) malloc(col * row * sizeof(float));
    deviceCopy_matrix = (float *) malloc(col * row * sizeof(float));
    for (int j = 0; j < row; j++) {
        for (int i = 0; i < col; i++) {
            host_matrix[j * col + i] = (j * col + i) + (j * col + i);
            printf("%2d,%2f ", (j * col + i) , host_matrix[j * col + i] );
        }
    }
    size_t pitch;
    cudaMallocPitch(&device_matrix, &pitch, col * sizeof(float), row);
    dim3 block;
    block.x = (1024/2);
    block.y = (1024/2);
    dim3 grid;
    grid.x = row / block.x;
    grid.y = col / block.y;
    kernel<<<grid, block>>>(device_matrix, pitch);
    cudaMemcpy2D(deviceCopy_matrix, col * sizeof(float), device_matrix, pitch, col * sizeof(float), row, cudaMemcpyDeviceToHost);
    verify(host_matrix, deviceCopy_matrix, col * row);
    free(host_matrix);
    cudaFree(device_matrix);
    free(deviceCopy_matrix);
}
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 8192
#define M 8192

__global__ void kernel(float * d_matrix, size_t pitch) {
    int colsPerThread = 8;//32 threads per block ,256 cells in block-> 256/32
    int rowstart = blockIdx.y * blockDim.y + (threadIdx.y * colsPerThread);
    for (int j = rowstart; j < rowstart+colsPerThread; j ++) {
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
      //  int colstart = blockIdx.x * blockDim.x + (threadIdx.x * colsPerThread);
      //  for (int i = colstart; i < colstart + colsPerThread; i ++) {
      //      row_d_matrix[i] = i;
      //  }
    }
}

void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        //printf("h: %f,d: %f ",h[i],d[i]);
        printf("%d",i);
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
__global__ void fill(float * d_matrix, size_t pitch) {
    int index ;
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < N; j += blockDim.y * gridDim.y) {
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
           // row_d_matrix[i] = (j * M + i) + (j * M + i);
            index = j * M + i;
            if (index <= M || (index % (M-1)) == 0){
                row_d_matrix[i] = 0.0;
            }
            else if (index >= 67100672){// might be one off
                row_d_matrix[i] = 100.0;
            }
        }
    }
   /* row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row == 400 ){
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
        col = colStartPos;
        for ( i = 0; i < colsPerThread; i++) {
            col += i;
            if (col < 331){
                ippos[i]=100.0;
                oppos[i]=100.0;
            }else{
                break;
            }
        }

    }
    else if (row == 200){
        col = colStartPos;
        if ( col <= 500 && (col + ncols) >= 500 ){
            ippos[500]=100.0;
            oppos[500]=100.0;   
        }
        
    }
*/
}


int main() {

    float *h_matrix;
    float *d_matrix;
    float *dc_matrix;

    h_matrix = (float *) malloc(M * N * sizeof(float));
    dc_matrix = (float *) malloc(M * N * sizeof(float));

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            h_matrix[j * M + i] = (j * M + i);
        }
    }

/*
    int index;
    for (int j = 0 ; j < N; j++ ) {
        for (int i = 0 ; i < M; i++) {
            index = j * M + i;
            if (index <= M || (index % (M-1)) == 0){
                h_matrix[j * M + i] = 0.0;
            }
            else if (index >= 67100672){// might be one off
                h_matrix[j * M + i] = 100.0;
            }
        }
    }
*/
    size_t pitch;
    hipMallocPitch(&d_matrix, &pitch, M * sizeof(float), N);

    //dim3 threadsPerBlock((1024/4), (1024/4), 1); // number of threads per block 256x256
    //dim3 numBlocks((N/threadsPerBlock.x),(M/threadsPerBlock.y), 1); // number of blocks in grid 32x32
    dim3 threadsPerBlock(32, 32, 1); // number of threads per block 
    dim3 numBlocks(N/threadsPerBlock.x,M/threadsPerBlock.y, 1); // number of blocks in grid 16x16

    kernel<<<numBlocks, threadsPerBlock>>>(d_matrix, pitch);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
        return 0;
    }
    hipMemcpy2D(dc_matrix, M * sizeof(float), d_matrix, pitch, M * sizeof(float), N, hipMemcpyDeviceToHost);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
        return 0;
    }
    verify(h_matrix, dc_matrix, M * N);

    free(h_matrix);
    hipFree(d_matrix);
    free(dc_matrix);
}