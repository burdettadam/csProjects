/*#include <stdio.h>
#include <assert.h>
#define row 22
#define col 22
__global__ void kernel(float * device_matrix, size_t pitch) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < row; j += blockDim.y * gridDim.y) {
        float* row_device_matrix = (float*)((char*)device_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < col; i += blockDim.x * gridDim.x) {
            row_device_matrix[i] = (j * col + i) + (j * col + i);
        }
    }
}
void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        printf("%2f ",d[i]);
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
int main() {
    float *host_matrix;
    float *device_matrix;
    float *deviceCopy_matrix;
    host_matrix = (float *) malloc(col * row * sizeof(float));
    deviceCopy_matrix = (float *) malloc(col * row * sizeof(float));
    for (int j = 0; j < row; j++) {
        for (int i = 0; i < col; i++) {
            host_matrix[j * col + i] = (j * col + i) + (j * col + i);
            printf("%2d,%2f ", (j * col + i) , host_matrix[j * col + i] );
        }
    }
    size_t pitch;
    cudaMallocPitch(&device_matrix, &pitch, col * sizeof(float), row);
    dim3 block;
    block.x = (1024/2);
    block.y = (1024/2);
    dim3 grid;
    grid.x = row / block.x;
    grid.y = col / block.y;
    kernel<<<grid, block>>>(device_matrix, pitch);
    cudaMemcpy2D(deviceCopy_matrix, col * sizeof(float), device_matrix, pitch, col * sizeof(float), row, cudaMemcpyDeviceToHost);
    verify(host_matrix, deviceCopy_matrix, col * row);
    free(host_matrix);
    cudaFree(device_matrix);
    free(deviceCopy_matrix);
}
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 160
#define M 160

__global__ void kernel(float * d_matrix, size_t pitch) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < N; j += blockDim.y * gridDim.y) {
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
            row_d_matrix[i] = (j * M + i) + (j * M + i);
        }
    }
}

void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}

int main() {

    float *h_matrix;
    float *d_matrix;
    float *dc_matrix;

    h_matrix = (float *) malloc(M * N * sizeof(float));
    dc_matrix = (float *) malloc(M * N * sizeof(float));

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            h_matrix[j * M + i] = (j * M + i) + (j * M + i);
        }
    }

    size_t pitch;
    hipMallocPitch(&d_matrix, &pitch, M * sizeof(float), N);

    dim3 grid((N/16),(M/16), 1); // number of blocks in grid
    dim3 block((1024/2), (1024/2), 1); // number of threads per block

    kernel<<<grid, block>>>(d_matrix, pitch);

    hipMemcpy2D(dc_matrix, M * sizeof(float), d_matrix, pitch, M * sizeof(float), N, hipMemcpyDeviceToHost);

    verify(h_matrix, dc_matrix, M * N);

    free(h_matrix);
    hipFree(d_matrix);
    free(dc_matrix);
}