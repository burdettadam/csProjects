#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#define row 11
#define col 11
__global__ void kernel(float * device_matrix, size_t pitch) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < row; j += blockDim.y * gridDim.y) {
        float* row_device_matrix = (float*)((char*)device_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < col; i += blockDim.x * gridDim.x) {
            row_device_matrix[i] = (j * col + i) + (j * col + i);
        }
    }
}
void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}
int main() {
    float *host_matrix;
    float *device_matrix;
    float *deviceCopy_matrix;
    host_matrix = (float *) malloc(col * row * sizeof(float));
    deviceCopy_matrix = (float *) malloc(col * row * sizeof(float));
    for (int j = 0; j < row; j++) {
        for (int i = 0; i < col; i++) {
            host_matrix[j * col + i] = (j * col + i) + (j * col + i);
            printf("%2d ",host_matrix)
        }
    }
    size_t pitch;
    hipMallocPitch(&device_matrix, &pitch, col * sizeof(float), row);
    dim3 block;
    block.x = row;
    block.y = col;
    dim3 grid;
    grid_size.x = row / block.x;
    grid_size.y = col / block.y;
    kernel<<<grid, block>>>(device_matrix, pitch);
    hipMemcpy2D(deviceCopy_matrix, col * sizeof(float), device_matrix, pitch, col * sizeof(float), row, hipMemcpyDeviceToHost);
    verify(host_matrix, deviceCopy_matrix, col * row);
    free(host_matrix);
    hipFree(device_matrix);
    free(deviceCopy_matrix);
}