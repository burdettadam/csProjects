#include "hip/hip_runtime.h"
// This example demonstrates how to launch two dimensional grids of CUDA threads.

#include <stdlib.h>
#include <stdio.h>
const int tableSize = 8192;

__global__ void kernel(int *array)
{
  // compute the two dimensional index of this particular
  // thread in the grid

  // do the usual computation separately in each dimension:
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  // use the two 2D indices to compute a single linear index
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;

  // use the two 2D block indices to compute a single linear block index
  int result = blockIdx.y * gridDim.x + blockIdx.x;

  // write out the result
 // array[index] = result;
  array[index] = index_x + (index_y * tableSize); // should print out index of each cell.
                                                 //                         N 
  int C = index_x + index_y*tableSize;           // node (i,j)              |
  int N = index_x + (index_y+1)*tableSize;       // node (i,j+1)            |
  int S = index_x + (index_y-1)*tableSize;       // node (i,j-1)     W ---- C ---- E
  int E = (index_x+1) + index_y*tableSize;       // node (i+1,j)            |
  int W = (index_x-1) + index_y*tableSize;       //                         |
                                                 //                         S 

}
__global__ void fillPlateWithTemperature( int *current,int *old)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;

    for (int row = 0 ; row < tableSize; row++ ) {
        for (int col = 0 ; col < tableSize; col++) {
            // the checks will slow you down alot....
            if (row == 0 || col == 0 || col == tableSize-1 ){
                current[row][col] = 0.0;
                old[row][col] = 0.0;
            }
            else if (row == (tableSize - 1) ){
                current[row][col] = 100.0;
                old[row][col] = 100.0;
            }
        }
    }
    for (int col = 0; col < 331 ; col++) {
        current[400][col]=100.0;
        old[400][col]=100.0;
    }
    current[200][500]=100.0;
    old[200][500]=100.0;


    
}


int main(void)
{
  int num_elements_x = tableSize;
  int num_elements_y = tableSize;

  int num_bytes = num_elements_x * num_elements_y * sizeof(int);

  int *device_array = 0;
  int *host_array = 0;

  // malloc a host array
  host_array = (int*)malloc(num_bytes);

  // hipMalloc a device array
  hipMalloc((void**)&device_array, num_bytes);

  // if either memory allocation failed, report an error message
  if(host_array == 0 || device_array == 0)
  {
    printf("couldn't allocate memory, host: %d, device: %d\n",host_array,device_array);
  //  return 1;
  }

  // choose a two dimensional launch configuration
  // use the dim3 type when launches are not one dimensional

  // create 4x4 thread blocks
  dim3 block_size;
  block_size.x = tableSize/4;
  block_size.y = tableSize/4;

  // configure a two dimensional grid as well
  dim3 grid_size;
  grid_size.x = num_elements_x / block_size.x;
  grid_size.y = num_elements_y / block_size.y;

  // grid_size & block_size are passed as arguments to the
  // triple chevrons as usual
  kernel<<<grid_size,block_size>>>(device_array);

  // download and inspect the result on the host:
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print out the result element by element
  for(int row = 0; row < num_elements_y; ++row)
  {
    for(int col = 0; col < num_elements_x; ++col)
    {
      printf("%2d ", host_array[row * num_elements_x + col]);
    }
    printf("\n");
  }
  printf("\n");

  // deallocate memory
  free(host_array);
  hipFree(device_array);
}