// This example demonstrates how to launch two dimensional grids of CUDA threads.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
const int tableSize = 8192;
const int blocksize = 1024;
__global__ void sumreduct(size_t pitch, float* dev_vector, float* dev_matrix, int  columns, int N)
{
int idx = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

while(idx<N)
{
    dev_vector[idx] = *(float *)( ((char*)dev_matrix + idx * pitch) + columns);
    idx += stride;
} 
}

__global__ void kernel(float *array, size_t pitch)
{
  // compute the two dimensional index of this particular
  // thread in the grid

  // do the usual computation separately in each dimension:
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  // use the two 2D indices to compute a single linear index
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;

  // use the two 2D block indices to compute a single linear block index
 // int result = blockIdx.y * gridDim.x + blockIdx.x;

  // write out the result
 // array[index] = result;
 // array[index] = index_x + (index_y * tableSize); // should print out index of each cell.
                                                 //                         N 
//  int C = index_x + index_y*tableSize;           // node (i,j)              |
//  int N = index_x + (index_y+1)*tableSize;       // node (i,j+1)            |
//  int S = index_x + (index_y-1)*tableSize;       // node (i,j-1)     W ---- C ---- E
//  int E = (index_x+1) + index_y*tableSize;       // node (i+1,j)            |
//  int W = (index_x-1) + index_y*tableSize;       //                         |
                                                 //                         S 
 // for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (blocksize / 2); i += blockDim.x * gridDim.x){
 //       y[i] = a * x[i] + y[i];
 //   }
  for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < tableSize; j += blockDim.y * gridDim.y) {
      float* row_d_matrix = (float*)((char*)array + j*pitch);
      for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < tableSize; i += blockDim.x * gridDim.x) {
          row_d_matrix[i] = i;
         // row_d_matrix[i] = (j * tableSize + i) + (j * tableSize + i);
      }
  }
}
/*
__global__ void fillPlateWithTemperature( int *current,int *old)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int grid_width = gridDim.x * blockDim.x;
  int index = index_y * grid_width + index_x;

    for (int row = 0 ; row < tableSize; row++ ) {
        for (int col = 0 ; col < tableSize; col++) {
            // the checks will slow you down alot....
            if (row == 0 || col == 0 || col == tableSize-1 ){
                current[row][col] = 0.0;
                old[row][col] = 0.0;
            }
            else if (row == (tableSize - 1) ){
                current[row][col] = 100.0;
                old[row][col] = 100.0;
            }
        }
    }
    for (int col = 0; col < 331 ; col++) {
        current[400][col]=100.0;
        old[400][col]=100.0;
    }
    current[200][500]=100.0;
    old[200][500]=100.0;


    
}
*/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }


int main(void)
{
  int num_elements_x = tableSize;
  int num_elements_y = tableSize;

  int num_bytes = num_elements_x * num_elements_y * sizeof(int);

  float *device_array = 0;
  float host_array [tableSize][tableSize];
  size_t pitch;
  // cudaMalloc a device array
  hipMallocPitch(&device_array, &pitch, tableSize * sizeof(float), tableSize * sizeof(float));
  hipMemcpy2D(device_array,pitch,host_array,tableSize * sizeof(float),tableSize * sizeof(float),tableSize, hipMemcpyHostToDevice);


  // create 512x512 thread blocks
  dim3 block_size;
  block_size.x = blocksize/2;
  block_size.y = blocksize/2;

  // configure a two dimensional grid as well
  dim3 grid_size;
  grid_size.x = iDivUp( num_elements_x , block_size.x);
  grid_size.y = iDivUp( num_elements_y , block_size.y);

  // grid_size & block_size are passed as arguments to the
  // triple chevrons as usual
  kernel<<<grid_size,block_size>>>( device_array, pitch );

  // download and inspect the result on the host:
  hipMemcpy2D(host_array, pitch , device_array,  tableSize * sizeof(float),tableSize * sizeof(float),tableSize, hipMemcpyDeviceToHost);
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print out the result element by element
  for(int row = 0; row < num_elements_y; ++row)
  {
    for(int col = 0; col < num_elements_x; ++col)
    {
      printf("%2d ", host_array[row * num_elements_x + col]);
    }
    printf("\n");
  }
  printf("\n");

  // deallocate memory
  free(host_array);
  hipFree(device_array);
}