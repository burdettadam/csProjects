#include "hip/hip_runtime.h"

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include<conio.h>
 
#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16
 
#define N 256
#define M 256
 

/*****************/
/* CUDA MEMCHECK */
/*****************/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{

   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) { getch(); exit(code); }
   }
}

 

/*******************/
/* iDivUp FUNCTION */
/*******************/

int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }



/******************/
/* TEST KERNEL 2D */
/******************/

__global__ void test_kernel_2D(float* d_a, size_t pitch)
{

int    tidx = blockIdx.x*blockDim.x+threadIdx.x;
int    tidy = blockIdx.y*blockDim.y+threadIdx.y;
 

if ((tidx<M) && (tidy<N))
   {
      float* row_a = (float*)((char*)d_a + tidx*pitch);
      row_a[tidy] = row_a[tidy] * row_a[tidy];
   }
}

 
/********/
/* MAIN */
/********/

int main()
{
   float a[N][M];
   float *d_a;
   size_t pitch;
 

for (int i=0; i<N; i++)
    for (int j=0; j<M; j++) {
         a[i][j] = 3.f;
         printf("row %i column %i value %f \n",i,j,a[i][j]);
     }


// --- 2D pitched allocation and host->device memcopy

   gpuErrchk(hipMallocPitch(&d_a,&pitch,M*sizeof(float),N));
 
   gpuErrchk(hipMemcpy2D(d_a,pitch,a,M*sizeof(float),M*sizeof(float),N,hipMemcpyHostToDevice));

   dim3 GridSize1(iDivUp(M,BLOCKSIZE_x),iDivUp(N,BLOCKSIZE_y));

   dim3 BlockSize1(BLOCKSIZE_y,BLOCKSIZE_x);

   test_kernel_2D<<<GridSize1,BlockSize1>>>(d_a,pitch);

   gpuErrchk(hipPeekAtLastError());

   gpuErrchk(hipDeviceSynchronize());

   gpuErrchk(hipMemcpy2D(a,M*sizeof(float),d_a,pitch,M*sizeof(float),N,hipMemcpyDeviceToHost));

   for (int i=0; i<N; i++) for (int j=0; j<M; j++) printf("row %i column %i value %f\n",i,j,a[i][j]);

   getch();

   return 0;

}